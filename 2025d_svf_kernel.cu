
#include <hip/hip_runtime.h>
#include <math.h>
#define PI 3.141592654

__device__ float Deg2Rad(float degree)
{
    return degree * PI / 180.0f;
}

__device__ float AnnulusWeight(float alt, float alt_interval, float azi_interval)
{
    /*
    Calculate the weighing factor of each sampling point in the RayTracing process.
    Based on the formula of the area of sphere segment (Area = (azimuth1 - azimuth2) * (cos(altitude1) - cos(altitude2))) divided by the total area of the hemisphere.
    
    Parameters:
        - alt: altitude of the sampling point in radians
        - alt_interval: altitude interval in radians
        - azi_interval: azimuth interval in radians
        - area_total: total area of the hemisphere

    Returns:
        - weighted_area_ray: the weighing factor of the sampling point
    */
   float area_ray = azi_interval * (cosf(alt - alt_interval/2.0f) - cosf(alt + alt_interval/2.0f)); 
   float weighted_area_ray = area_ray * cosf(alt);

   return weighted_area_ray;
}

__global__ void svfcalculator(float *svf_out, float *svfE_out, float *svfS_out, float *svfW_out, float *svfN_out,
                              float *svfveg_out, float *svfEveg_out, float *svfSveg_out, float *svfWveg_out, float *svfNveg_out,
                              float *svfaveg_out, float *svfEaveg_out, float *svfSaveg_out, float *svfWaveg_out, float *svfNaveg_out,
                              float *dsm, float *cdsm, float *tdsm,
                              float scale, int width, int height,
                              float traceRadius, float azimuth_start, float azimuth_end, float azimuth_interval, float altitude_interval)
{
        /**
     * Calculate Sky View Factor (SVF) using ray tracing on a GPU.
     *
     * This kernel function calculates the SVF for each pixel in the input DSM (Digital Surface Model) and vegetation models.
     * The SVF is calculated for different azimuth and altitude angles, and the results are stored in the output arrays.
     *
     * Args:
     *     svf_out (float*): Output array for overall SVF.
     *     svfE_out (float*): Output array for eastward SVF.
     *     svfS_out (float*): Output array for southward SVF.
     *     svfW_out (float*): Output array for westward SVF.
     *     svfN_out (float*): Output array for northward SVF.
     *     svfveg_out (float*): Output array for overall vegetation SVF.
     *     svfEveg_out (float*): Output array for eastward vegetation SVF.
     *     svfSveg_out (float*): Output array for southward vegetation SVF.
     *     svfWveg_out (float*): Output array for westward vegetation SVF.
     *     svfNveg_out (float*): Output array for northward vegetation SVF.
     *     svfaveg_out (float*): Output array for overall adjusted vegetation SVF.
     *     svfEaveg_out (float*): Output array for eastward adjusted vegetation SVF.
     *     svfSaveg_out (float*): Output array for southward adjusted vegetation SVF.
     *     svfWaveg_out (float*): Output array for westward adjusted vegetation SVF.
     *     svfNaveg_out (float*): Output array for northward adjusted vegetation SVF.
     *     dsm (float*): Input DSM array.
     *     cdsm (float*): Input vegetation height model array.
     *     tdsm (float*): Input vegetation canopy bottom height model array.
     *     scale (float): Scale factor for the height.
     *     width (int): Width of the input arrays.
     *     height (int): Height of the input arrays.
     *     traceRadius (float): The radius for ray tracing.
     *     azimuth_start (float): Starting azimuth angle in degrees.
     *     azimuth_end (float): Ending azimuth angle in degrees.
     *     azimuth_interval (float): Azimuth interval in degrees.
     *     altitude_interval (float): Altitude interval in degrees.
     *
     * Returns:
     *     svf's, svfveg's and svfaveg's are stored in the output arrays.
     */
    //Calculate the index of the current thread and avoid out-of-bounds access
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    if (ix >= width || iy >= height)
        return;
    int index = ix + iy * width;

    //Read the DSM values at the current index
    float clr00 = dsm[index];

    // Initialize the SVF arrays
    float svf = 0.0f;
    float svfE = 0.0f;
    float svfS = 0.0f;
    float svfW = 0.0f;
    float svfN = 0.0f;

    float svfveg = 0.0f;
    float svfEveg = 0.0f;
    float svfSveg = 0.0f;
    float svfWveg = 0.0f;
    float svfNveg = 0.0f;

    float svfaveg = 0.0f;
    float svfEaveg = 0.0f;
    float svfSaveg = 0.0f;
    float svfWaveg = 0.0f;
    float svfNaveg = 0.0f;

    //Initialize the angle parameters
    float alt_start = Deg2Rad(0.0f);
    float alt_end = Deg2Rad(90.0f);
    float alt_interval = Deg2Rad(altitude_interval);

    float azi_start = Deg2Rad(0.0f);
    float azi_end = Deg2Rad(360.0f);
    float azi_interval = Deg2Rad(azimuth_interval);

    float total_ray_area = 0.0f;
    float total_ray_areaE = 0.0f;
    float total_ray_areaS = 0.0f;
    float total_ray_areaW = 0.0f;
    float total_ray_areaN = 0.0f;

    //Main calculation loops for the SVFs
    for (float alt = alt_start; alt <= alt_end; alt += alt_interval)
    {
        float weight   = AnnulusWeight(alt, alt_interval, azi_interval);

        for (float azimuth = azi_start; azimuth < azi_end; azimuth += azi_interval)
        {
            float radius = 1.0f;
            float cos_azi = cosf(azimuth);
            float sin_azi = sinf(azimuth);
            bool crossed_veg = false;
            bool crossed_dsm = false;

            total_ray_area += weight;

            if(azimuth >= 0.0f && azimuth <= PI){
                total_ray_areaS += weight;
            } if (azimuth >= (0.5f * PI) && azimuth <= (1.5f * PI)){
                total_ray_areaW += weight;
            } if (azimuth >= (1.0f * PI) && azimuth <= (2.0f * PI)){
                total_ray_areaN += weight;
            } if (azimuth >= (1.5f * PI) || azimuth <= (0.5f * PI)){
                total_ray_areaE += weight;
            }            

            // Trace the ray until traceRadius
            while (radius <= traceRadius)
            {
                float dx = cos_azi;
                float dy = sin_azi;
                float x_float = float(ix);
                float y_float = float(iy);
                int x = int(roundf(x_float + radius * cosf(alt) * dx));
                int y = int(roundf(y_float + radius * cosf(alt) * dy));               
                if (x < 0 || x >= width || y < 0 || y >= height)
                    break;
                int index2 = x + y * width;

                float dsm_height = dsm[index2];
                float veg_height = cdsm[index2];
                float canopy_bottom_height = tdsm[index2];
                float ray_height = clr00 + radius/scale * sinf(alt);

                //Avegetation SVF calculations + anisotropics
                if (ray_height < dsm_height && crossed_veg && !crossed_dsm){
                    svfaveg += weight;
                    svfveg -= weight;
                    if(azimuth >= 0.0f && azimuth <= PI){
                        svfSaveg += weight;
                        svfSveg -= weight;
                    }
                    if(azimuth >= (0.5f * PI) && azimuth <= (1.5f * PI)){
                        svfWaveg += weight;
                        svfWveg -= weight;
                    }
                    if(azimuth >= (1.0f * PI) && azimuth <= (2.0f * PI)){
                        svfNaveg += weight;
                        svfNveg -= weight;
                    }
                    if(azimuth >= (1.5f * PI) || azimuth <= (0.5f * PI)){
                        svfEaveg += weight ;
                        svfEveg -= weight;
                    }
                }

                //Regular SVF calculations + anisotropics
                if (ray_height < dsm_height && !crossed_dsm)
                {
                    svf += weight;                    
                    if(azimuth >= 0.0f && azimuth <= PI){
                        svfS += weight;
                    }
                   if(azimuth >= (0.5f * PI) && azimuth <= (1.5f * PI)){
                        svfW += weight;
                    }
                    if(azimuth >= (1.0f * PI) && azimuth <= (2.0f * PI)){
                        svfN += weight;
                    }
                    if(azimuth >= (1.5f * PI) || azimuth <= (0.5f * PI)){
                        svfE += weight;
                    }
                    crossed_dsm = true;
                }

                //Vegetation SVF calculations + anisotropics
                if (ray_height < veg_height && ray_height > canopy_bottom_height && !crossed_dsm && !crossed_veg){
                    svfveg += weight;
                    if(azimuth >= 0.0f && azimuth <= PI){
                        svfSveg += weight;
                    }
                   if(azimuth >= (0.5f * PI) && azimuth <= (1.5f * PI)){
                        svfWveg += weight;
                    }
                    if(azimuth >= (1.0f * PI) && azimuth <= (2.0f * PI)){
                        svfNveg += weight;
                    }
                    if(azimuth >= (1.5f * PI) || azimuth <= (0.5f * PI)){
                        svfEveg += weight;
                    }
                    crossed_veg = true;
                }

                if(crossed_dsm && crossed_veg){
                    break;
                }

                // Adjust step size based on altitude angle
                float step_size = fmaxf(1.0f, (radius * cosf(alt)) * 0.1f);
                radius += step_size;
            }
        }
    }

    // Write the result to the output
    svf_out[index]  = 1.0f - svf/total_ray_area;
    svfE_out[index] = 1.0f - svfE/total_ray_areaE;
    svfS_out[index] = 1.0f - svfS/total_ray_areaS;
    svfW_out[index] = 1.0f - svfW/total_ray_areaW;
    svfN_out[index] = 1.0f - svfN/total_ray_areaN;

    svfveg_out[index]  = 1.0f - svfveg/total_ray_area;
    svfEveg_out[index] = 1.0f - svfEveg/total_ray_areaE;
    svfSveg_out[index] = 1.0f - svfSveg/total_ray_areaS;
    svfWveg_out[index] = 1.0f - svfWveg/total_ray_areaW;
    svfNveg_out[index] = 1.0f - svfNveg/total_ray_areaN;

    svfaveg_out[index]  = 1.0f - svfaveg/total_ray_area;
    svfEaveg_out[index] = 1.0f - svfEaveg/total_ray_areaE;
    svfSaveg_out[index] = 1.0f - svfSaveg/total_ray_areaS;
    svfWaveg_out[index] = 1.0f - svfWaveg/total_ray_areaW;
    svfNaveg_out[index] = 1.0f - svfNaveg/total_ray_areaN;
}